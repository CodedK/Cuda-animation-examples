#include "hip/hip_runtime.h"
#include "includes/general.h"
#include "includes/cpu_bitmap.h"
#include "includes/gpu_anim.h"
#define DIM 1000




// modify like this in hipComplex structure
// hipComplex( float a, float b ) : r(a), i(b)  {}  -->  __device__ hipComplex( float a, float b ) : r(a), i(b)  {}
struct hipComplex {
	float   r;
	float   i;
	__device__ hipComplex( float a, float b ) : r(a), i(b)  {}
	__device__ float magnitude2( void ) {
		return r * r + i * i;
		// return sqrt(r * r + i * i);
	}
	__device__ hipComplex operator*(const hipComplex& a) {
		return hipComplex(r*a.r - i*a.i, i*a.r + r*a.i);
	}
	__device__ hipComplex operator+(const hipComplex& a) {
		return hipComplex(r+a.r, i+a.i);
	}
	__device__ hipComplex operator-(const hipComplex& a) {
		return hipComplex(r-a.r, i-a.i);
	}
};
__device__ int julia( int x, int y, int ticks ) {
	// float euler = 2.718281;
	float scale =  1.5;
	// float scale =  0.5 + 1/(ticks*0.05);

	float jx = scale * (float)(DIM/2 - x)/(DIM/2);
	float jy = scale * (float)(DIM/2 - y)/(DIM/2);

	// float step =0;
	// float relu =0;
	// step = ticks % 12;
	// relu = logf(1+powf(euler, (step - 13) ));

	// c = 1j # dentrite fractal
	// c = -0.87 + 0.156 # Julia set
	// c = -0.123 + 0.745j # douady's rabbit fractal
	// c = -0.750 + 0j # san marco fractal
	// c = -0.391 - 0.587j # siegel disk fractal
	// c = -0.7 - 0.3j # NEAT cauliflower thingy
	// c = -0.75 - 0.2j # galaxies
	// c = -0.75 + 0.15j # groovy
	// c = -0.7 + 0.35j # frost

	// # JULIA
	// float julia = cosf(ticks*0.01)*0.23; // (ticks*slow_time) * fluctuation between -0.23 and 0.23
	// hipComplex c(-0.87, julia); // for x -0.87, max y:0.23

	// # DOUADY
	// float douady = cosf(ticks*0.01)*0.977; // (ticks*slow_time) * fluctuation between -0.977 and 0.977
	// hipComplex c(-0.123, douady); // for x -0.123, max y:0.977

	// # SIEGEL
	float siegel = cosf(ticks*0.1)*0.709; // (ticks*slow_time) * fluctuation between -0.709 and 0.709
	hipComplex c(-0.391, siegel); // for x -0.391, max y:0.709



	hipComplex a(jx, jy);
	int i = 0;
	for (i=0; i<500; i++) {
		a = a * a * c;
		if (a.magnitude2() > 1000)
			return 0;
	}
	return 1;
}

__global__ void kernel( unsigned char *ptr, int ticks ) {
	// map from blockIdx to pixel position
	int x = blockIdx.x;
	int y = blockIdx.y;
	int offset = x + y * gridDim.x;

	// unsigned char grey = (unsigned char)(128.0f + 127.0f * cos(x/10.0f - ticks/7.0f) / (y/10.0f + 1.0f));
	// now calculate the value at that position
	int juliaValue = julia( x, y, ticks );
	ptr[offset*4 + 0] = 255 * juliaValue;
	ptr[offset*4 + 1] = 0;
	ptr[offset*4 + 2] = 0;
	ptr[offset*4 + 3] = 255;
}
// globals needed by the update routine
struct DataBlock {
	unsigned char   *dev_bitmap;
};


void generate_frame( unsigned char *pixels, void*, int ticks ) {
    dim3    grid(DIM,DIM);
    // dim3    threads(16,16);
    kernel<<<grid,1>>>( pixels, ticks );
    // printf("%f\n", ticks);
}

int main( void ) {
    GPUAnimBitmap  bitmap( DIM, DIM, NULL );
    bitmap.anim_and_exit(
        (void (*)(uchar4*,void*,int)) generate_frame, NULL
    );
}



// int main( void ) {
// 	int ticks;
// 	DataBlock   data;
// 	CPUBitmap bitmap( DIM, DIM, &data );
// 	unsigned char    *dev_bitmap;
// 	HANDLE_ERROR( hipMalloc( (void**)&dev_bitmap, bitmap.image_size() ) );
// 	data.dev_bitmap = dev_bitmap;
// 	dim3    grid(DIM,DIM);
// 	// dim3    grid(DIM/16,DIM/16);
// 	// dim3    threads(16,16);
// 	ticks=0;
// 	kernel<<<grid,1>>>( dev_bitmap, ticks );
// 	// kernel<<<grid,1>>>( dev_bitmap );
// 	HANDLE_ERROR( hipMemcpy( bitmap.get_ptr(), dev_bitmap,
// 							  bitmap.image_size(),
// 							  hipMemcpyDeviceToHost ) );
// 	HANDLE_ERROR( hipFree( dev_bitmap ) );
// 	bitmap.display_and_exit();
// }
