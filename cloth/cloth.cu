#include "hip/hip_runtime.h"
#include "includes/general.h"
#include "includes/gpu_anim.h"
#define DIM 1024
__global__ void kernel( uchar4 *ptr, int ticks ) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;
	// now calculate the value at that position
	float fx = x - DIM/2;
	float fy = y - DIM/2;
	// Moving cloth
	float d = sqrtf(sqrtf( fx * fx + fy * fy));
	unsigned char grey = (unsigned char) (128.0f+127.0f*cos(d/20.0f - ticks/7.0f) / (d/20.0f + 1.0f));
	ptr[offset].x = grey - x%ticks;
	ptr[offset].y = grey - y%ticks;
	ptr[offset].z = grey % ticks;
	ptr[offset].w = 50%ticks;
}
void generate_frame( uchar4 *pixels, void*, int ticks ) {
	dim3    grids(DIM/32,DIM/32);
	dim3    threads(32,32);
	kernel<<<grids,threads>>>( pixels, ticks );
}
int main( void ) {
	GPUAnimBitmap  bitmap( DIM, DIM, NULL );
	bitmap.anim_and_exit( (void (*)(uchar4*,void*,int))generate_frame, NULL );
}
